
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <memory>

template <typename action_type, typename filter_type>
__global__ void transform_and_count (
    const int n,
    const int *data,
    int *result,
    action_type action,
    filter_type filter)
{
  const int i = threadIdx.x + blockIdx.x * blockDim.x;

  if (i < n && filter (action (data[i])))
    atomicAdd (result, 1);
}


int main ()
{
  const int n = 10'000'000;
  std::unique_ptr<int[]> cpu_data (new int[n]);
  std::iota (cpu_data.get (), cpu_data.get () + n, 0);

  int *gpu_data {};
  hipMalloc (&gpu_data, n * sizeof (int));
  hipMemcpy (gpu_data, cpu_data.get (), n * sizeof (int), hipMemcpyHostToDevice);
  
  int *gpu_result {};
  hipMalloc (&gpu_result, sizeof (int));
  hipMemset (gpu_result, sizeof (int), 0);

  hipEvent_t begin, end;
  hipEventCreate (&begin);
  hipEventCreate (&end);

  hipEventRecord (begin);
  transform_and_count<<<(n + 127) / 128, 128>>> (
      n, gpu_data, gpu_result, 
      [] __device__ (int value) { return value * value; },
      [] __device__ (int value) { return value == 10'000'000; });
  hipEventRecord (end);

  int cpu_result {};
  hipMemcpy (&cpu_result, gpu_result, sizeof (int), hipMemcpyDeviceToHost);

  hipEventSynchronize (end);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, begin, end);

  std::cout << cpu_result << " (complete in " << milliseconds << ")\n";
}
