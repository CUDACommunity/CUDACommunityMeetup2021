
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <numeric>
#include <memory>
#include <array>

void chck (hipError_t status)
{
  if (status != hipSuccess)
    throw std::runtime_error ("Error!");
}

__global__ void kernel (
    int n,
    volatile int * flag, 
    int * data, 
    int *result)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  if (bid == 0)
    {
      if (tid == 0)
        {
          for (int i = 0; i < n; i++)
            data[i] = i + 1;

          *flag = 1;
        }
    }
  else
    {
      while (*flag == 0);

      for (int i = 0; i < n; i++)
        if (data[i] == 0)
          *result = 0;
    }
}

int main ()
{
  int *data {};
  int *r {};

  constexpr int max_n = 9;
  std::array<int, max_n> stat;
  stat.fill (0);

  for (int iteration = 0; iteration < 10000; iteration++)
    {
      hipMalloc (&r, sizeof (int));

      for (int n = 1; n < max_n; n++)
        {
          hipMalloc (&data, (n + 1) * sizeof (int));
          hipMemset (data, (n + 1) * sizeof (int), 0);
          hipMemset (r, sizeof (int), 1);

          kernel<<<1024, 32>>> (n, data + 0, data + 1, r);

          chck (hipFree (data));

          int cpu_r {};
          chck (hipMemcpy (&cpu_r, r, sizeof (int), hipMemcpyDeviceToHost));

          if (cpu_r == 0)
          {
            std::cerr << "fail\n";
            stat[n]++;
          }
        }

      hipFree (r);
    }

  for (int i = 1; i < max_n; i++)
      std::cout << "n=" << i << ": " << stat[i] << "\n";

  return 0;
}
