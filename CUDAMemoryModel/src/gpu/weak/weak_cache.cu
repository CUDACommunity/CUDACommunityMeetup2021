
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <memory>

__global__ void kernel (
    volatile int * const __restrict__ x, 
    int * const __restrict__ y, int *result)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  if (bid == 0 && tid == 0)
    {
      *y = 2;
      *x = 1;
    }
  else
    {
      int reg = *y;
      while (*x == reg);

      if (*y == 0)
        *result = 0;
    }
}

int main ()
{
  int *x {};
  int *y {};
  int *r {};

  hipMalloc (&r, sizeof (int));
  hipMalloc (&x, sizeof (int));
  hipMalloc (&y, sizeof (int));

  hipMemset (x, sizeof (int), 0);
  hipMemset (y, sizeof (int), 0);
  hipMemset (r, sizeof (int), 1);

  kernel<<<2056, 1024>>> (x, y, r);

  int cpu_r {};
  hipMemcpy (&cpu_r, r, sizeof (int), hipMemcpyDeviceToHost);

  std::cout << "passed = " << cpu_r << "\n";

  return 0;
}
