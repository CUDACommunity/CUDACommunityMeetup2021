#include "not_interesting.cuh"
#include <memory>

int main()
{
	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	int numElements = 50000;
	size_t size = numElements * sizeof(float);

	// Note: C++14
	auto h_A = std::make_unique<float[]>(numElements);
	auto h_B = std::make_unique<float[]>(numElements);
	auto h_C = std::make_unique<float[]>(numElements);

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A.get()[i] = rand()/(float)RAND_MAX;
		h_B.get()[i] = rand()/(float)RAND_MAX;
	}

	// Allocate the device input vector A
	float *d_A = NULL;
	err = hipMalloc((void **)&d_A, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device input vector B
	float *d_B = NULL;
	err = hipMalloc((void **)&d_B, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate the device output vector C
	float *d_C = NULL;
	err = hipMalloc((void **)&d_C, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the host input vectors A and B in host memory to the device input vectors in
	// device memory
	err = hipMemcpy(d_A, h_A.get(), size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(d_B, h_B.get(), size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
	vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	err = hipMemcpy(h_C.get(), d_C, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	verify_correctness(h_A.get(), h_B.get(), h_C.get(), numElements);

	// Free device global memory
	err = hipFree(d_A);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_B);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipFree(d_C);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

